#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include "npp.h"
#include <math.h>
#include <windows.h>
#include <chrono>
#include <cstdint>
#include <algorithm>
#include ""

#define VC_EXTRALEAN
#define WIN32_LEAN_AND_MEAN

// CUDA error checking Macro.
#define CUDA_CALL(x,y) {if((x) != hipSuccess){ \
  printf("CUDA error at %s:%d\n",__FILE__,__LINE__); \
  printf("  %s\n", hipGetErrorString(hipGetLastError())); \
  exit(EXIT_FAILURE);}\
  else{printf("CUDA Success at %d. (%s)\n",__LINE__,y); }}

double PCFreq = 0.0;
__int64 CounterStart = 0;

void StartCounter()
{
	LARGE_INTEGER li;
	if (!QueryPerformanceFrequency(&li))
		std::cout << "QueryPerformanceFrequency failed!\n";

	PCFreq = double(li.QuadPart) / 1000.0;

	QueryPerformanceCounter(&li);
	CounterStart = li.QuadPart;
}
double GetCounter()
{
	LARGE_INTEGER li;
	QueryPerformanceCounter(&li);
	return double(li.QuadPart - CounterStart) / PCFreq;
}

// Function Protypes.
unsigned int *
LoadPGM(char * sFileName, int & nWidth, int & nHeight, int & nMaxGray);

__global__ void
TransformKernel(const hipTextureObject_t d_img_tex, const float gxs, const float gys, uint8_t* __restrict const d_out, const int neww);

void InterpolateSum(const hipTextureObject_t d_img_tex, const int oldw, const int oldh, uint8_t* __restrict const d_out, const uint32_t neww, const uint32_t newh);

int main()
{

	auto image = new uint8_t[4];
	image[0] = 255;
	image[1] = 255;
	image[2] = 0;
	image[3] = 0;

	constexpr int oldw = 2;
	constexpr int oldh = 2;
	constexpr int neww = static_cast<int>(static_cast<double>(oldw) * 400.0);
	constexpr int newh = static_cast<int>(static_cast<double>(oldh) * 1000.0);
	const size_t total = static_cast<size_t>(neww)*static_cast<size_t>(newh);


	hipDeviceSetCacheConfig(hipFuncCachePreferL1);
	hipDeviceSetSharedMemConfig(hipSharedMemBankSizeFourByte);

	hipChannelFormatDesc chandesc_img = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
	hipArray* d_img_arr;
	CUDA_CALL(hipMallocArray(&d_img_arr, &chandesc_img, oldw, oldh, hipArrayTextureGather),"Memory Allocation.");
	CUDA_CALL(hipMemcpyToArray(d_img_arr, 0, 0, image, oldh * oldw, hipMemcpyHostToDevice), "Memory Cpoied to Array.");
	struct hipResourceDesc resdesc_img;
	memset(&resdesc_img, 0, sizeof(resdesc_img));
	resdesc_img.resType = hipResourceTypeArray;
	resdesc_img.res.array.array = d_img_arr;
	struct hipTextureDesc texdesc_img;
	memset(&texdesc_img, 0, sizeof(texdesc_img));
	texdesc_img.addressMode[0] = hipAddressModeClamp;
	texdesc_img.addressMode[1] = hipAddressModeClamp;
	texdesc_img.readMode = hipReadModeNormalizedFloat;
	texdesc_img.filterMode = hipFilterModePoint;
	texdesc_img.normalizedCoords = 0;
	hipTextureObject_t d_img_tex = 0;
	CUDA_CALL(hipCreateTextureObject(&d_img_tex, &resdesc_img, &texdesc_img, nullptr),"Texture Object Created.");

	uint8_t* d_out = nullptr;
	CUDA_CALL(hipMalloc(&d_out, total),"Memory Allocated.");
	StartCounter();
	InterpolateSum(d_img_tex, oldw, oldh, d_out, neww, newh);
	std::cout << GetCounter() << std::endl;
	auto h_out = new uint8_t[neww * newh];
	CUDA_CALL(hipMemcpy(h_out, d_out, total, hipMemcpyDeviceToHost),"Memory Copied.");

	std::cout << "Input stats: " << oldh << " rows, " << oldw << " cols." << std::endl;
	std::cout << "Output stats: " << newh << " rows, " << neww << " cols." << std::endl;
	getchar();
}


__global__ void
TransformKernel(const hipTextureObject_t d_img_tex, const float gxs, const float gys, uint8_t* __restrict const d_out, const int neww) {
	uint32_t x = (blockIdx.x << 9) + (threadIdx.x << 1);
	const uint32_t y = blockIdx.y;
	const float fy = (y + 0.5f)*gys - 0.5f;
	const float wt_y = fy - floor(fy);
	const float invwt_y = 1.0f - wt_y;
#pragma unroll
	for (int i = 0; i < 2; ++i, ++x) {
		const float fx = (x + 0.5f)*gxs - 0.5f;
		const float4 f = tex2Dgather<float4>(d_img_tex, fx + 0.5f, fy + 0.5f);
		const float wt_x = fx - floor(fx);
		const float invwt_x = 1.0f - wt_x;
		const float xa = invwt_x*f.w + wt_x*f.z;
		const float xb = invwt_x*f.x + wt_x*f.y;
		const float res = 255.0f*(invwt_y*xa + wt_y*xb) + 0.5f;
		// -----------------
		if (x < neww) d_out[y*neww + x] = res;
	}
}

void InterpolateSum(const hipTextureObject_t d_img_tex, const int oldw, const int oldh, uint8_t* __restrict const d_out, const uint32_t neww, const uint32_t newh) {
	const float gxs = static_cast<float>(oldw) / static_cast<float>(neww);
	const float gys = static_cast<float>(oldh) / static_cast<float>(newh);
	TransformKernel << <{((neww - 1) >> 9) + 1, newh}, 256 >> >(d_img_tex, gxs, gys, d_out, neww);
	hipDeviceSynchronize();
}

// Disable reporting warnings on functions that were marked with deprecated.
#pragma warning( disable : 4996 )

// Load PGM file.
unsigned int *
LoadPGM(char * sFileName, int & nWidth, int & nHeight, int & nMaxGray)
{
	char aLine[256];
	FILE * fInput = fopen(sFileName, "r");
	if (fInput == 0)
	{
		perror("Cannot open file to read");
		exit(EXIT_FAILURE);
	}
	// First line: version
	fgets(aLine, 256, fInput);
	std::cout << "\tVersion: " << aLine;
	// Second line: comment
	fgets(aLine, 256, fInput);
	std::cout << "\tComment: " << aLine;
	fseek(fInput, -1, SEEK_CUR);
	// Third line: size
	fscanf(fInput, "%d", &nWidth);
	std::cout << "\tWidth: " << nWidth;
	fscanf(fInput, "%d", &nHeight);
	std::cout << " Height: " << nHeight << std::endl;
	// Fourth line: max value
	fscanf(fInput, "%d", &nMaxGray);
	std::cout << "\tMax value: " << nMaxGray << std::endl;
	while (getc(fInput) != '\n');
	// Following lines: data
	unsigned int * pSrc_Host = new unsigned int[nWidth * nHeight];
	for (int i = 0; i < nHeight; ++i)
		for (int j = 0; j < nWidth; ++j)
			pSrc_Host[i*nWidth + j] = fgetc(fInput);
	fclose(fInput);

	return pSrc_Host;
}


