#include "hip/hip_runtime.h"

#include <stdio.h>
#include <windows.h>
#include "cuda_helper.h"
#include "timer_helper.h"
#include "pgm_helper.h"

// Function Protypes.
uint8_t *
LoadPGM(char * sFileName, int & nWidth, int & nHeight, int & nMaxGray);

void
WritePGM(char * sFileName, uint8_t * pDst_Host, int nWidth, int nHeight, int nMaxGray);

__global__ void
TransformKernel(const hipTextureObject_t d_img_texA, const hipTextureObject_t d_img_texB, const hipTextureObject_t d_img_texC, 
				const float gxs, const float gys, 
				const float gxsB, const float gysB,
				const float gxsC, const float gysC, 
				uint8_t* __restrict const d_out, const int neww);

void InterpolateSum(const hipTextureObject_t d_img_texA, const hipTextureObject_t d_img_texB, const hipTextureObject_t d_img_texC, const int oldw, const int oldh, uint8_t* __restrict const d_out, const uint32_t neww, const uint32_t newh);

int main()
{
#pragma region Variable Declaritions
	// Host parameter declarations.	
	int   nWidth, nHeight, nMaxGray;
	// Device parameter declaration.
	uint8_t* pDst_Dev = nullptr;
#pragma endregion

#pragma region Load image to the host
	std::cout << "Loading PGM file." << std::endl;
	auto pSrc_HostB = LoadPGM((char *)"./data/lena_beforeB.pgm", nWidth, nHeight, nMaxGray);
	auto pSrc_HostC = LoadPGM((char *)"./data/lena_beforeC.pgm", nWidth, nHeight, nMaxGray);
	auto pSrc_HostA = LoadPGM((char *)"./data/lena_beforeA.pgm", nWidth, nHeight, nMaxGray);
#pragma endregion

#pragma region Size Parameter Definitions
	int initial_width = nWidth;
	int initial_height = nHeight;
	int final_width = initial_width * 8;
	int final_heigth = initial_height * 8;
	size_t total = final_width*final_heigth;
#pragma endregion

	//Channel Description
	hipChannelFormatDesc chandesc_img = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);

#pragma region Array-Resource-Texture For Image A 
	hipArray* d_img_arrA;
	CUDA_CALL(hipMallocArray(&d_img_arrA, &chandesc_img, initial_width, initial_height, hipArrayTextureGather), "Memory Allocation.");
	CUDA_CALL(hipMemcpyToArray(d_img_arrA, 0, 0, pSrc_HostA, initial_width * initial_height, hipMemcpyHostToDevice), "Memory Cpoied to Array.");

	struct hipResourceDesc resdesc_imgA;
	memset(&resdesc_imgA, 0, sizeof(resdesc_imgA));
	resdesc_imgA.resType = hipResourceTypeArray;
	resdesc_imgA.res.array.array = d_img_arrA;

	struct hipTextureDesc texdesc_imgA;
	memset(&texdesc_imgA, 0, sizeof(texdesc_imgA));
	texdesc_imgA.addressMode[0] = hipAddressModeClamp;
	texdesc_imgA.addressMode[1] = hipAddressModeClamp;
	texdesc_imgA.readMode = hipReadModeNormalizedFloat;
	texdesc_imgA.filterMode = hipFilterModePoint;
	texdesc_imgA.normalizedCoords = 0;

	hipTextureObject_t d_img_texA = 0;
	CUDA_CALL(hipCreateTextureObject(&d_img_texA, &resdesc_imgA, &texdesc_imgA, nullptr), "Texture Object A Created.");
#pragma endregion

#pragma region Array-Resource-Texture For Image B
	hipArray* d_img_arrB;
	CUDA_CALL(hipMallocArray(&d_img_arrB, &chandesc_img, initial_width/2, initial_height/2, hipArrayTextureGather), "Memory Allocation.");
	CUDA_CALL(hipMemcpyToArray(d_img_arrB, 0, 0, pSrc_HostB, initial_width * initial_height / 4, hipMemcpyHostToDevice), "Memory Cpoied to Array.");

	struct hipResourceDesc resdesc_imgB;
	memset(&resdesc_imgB, 0, sizeof(resdesc_imgB));
	resdesc_imgB.resType = hipResourceTypeArray;
	resdesc_imgB.res.array.array = d_img_arrB;

	struct hipTextureDesc texdesc_imgB;
	memset(&texdesc_imgB, 0, sizeof(texdesc_imgB));
	texdesc_imgB.addressMode[0] = hipAddressModeClamp;
	texdesc_imgB.addressMode[1] = hipAddressModeClamp;
	texdesc_imgB.readMode = hipReadModeNormalizedFloat;
	texdesc_imgB.filterMode = hipFilterModePoint;
	texdesc_imgB.normalizedCoords = 0;

	hipTextureObject_t d_img_texB = 0;
	CUDA_CALL(hipCreateTextureObject(&d_img_texB, &resdesc_imgB, &texdesc_imgB, nullptr), "Texture Object B Created.");
#pragma endregion

#pragma region Array-Resource-Texture For Image C
	hipArray* d_img_arrC;
	CUDA_CALL(hipMallocArray(&d_img_arrC, &chandesc_img, initial_width/4, initial_height/4, hipArrayTextureGather), "Memory Allocation.");
	CUDA_CALL(hipMemcpyToArray(d_img_arrC, 0, 0, pSrc_HostC, initial_width * initial_height / 16, hipMemcpyHostToDevice), "Memory Copied to Array.");

	struct hipResourceDesc resdesc_imgC;
	memset(&resdesc_imgC, 0, sizeof(resdesc_imgC));
	resdesc_imgC.resType = hipResourceTypeArray;
	resdesc_imgC.res.array.array = d_img_arrC;

	struct hipTextureDesc texdesc_imgC;
	memset(&texdesc_imgC, 0, sizeof(texdesc_imgC));
	texdesc_imgC.addressMode[0] = hipAddressModeClamp;
	texdesc_imgC.addressMode[1] = hipAddressModeClamp;
	texdesc_imgC.readMode = hipReadModeNormalizedFloat;
	texdesc_imgC.filterMode = hipFilterModePoint;
	texdesc_imgC.normalizedCoords = 0;

	hipTextureObject_t d_img_texC = 0;
	CUDA_CALL(hipCreateTextureObject(&d_img_texC, &resdesc_imgC, &texdesc_imgC, nullptr), "Texture Object B Created.");
#pragma endregion

	//Device Output Memory Ops
	CUDA_CALL(hipMalloc(&pDst_Dev, total), "Memory Allocated for Device Output.");

	FastTimer ft; //Timer 
	ft.StartCounter();
	InterpolateSum(d_img_texA, d_img_texB, d_img_texC, initial_width, initial_height, pDst_Dev, final_width, final_heigth);
	std::cout << "Process finished in " << ft.GetCounter() << " ms." << std::endl;

	//Device Output Memory Ops
	auto pDst_Host = new uint8_t[final_width * final_heigth];
	CUDA_CALL(hipMemcpy(pDst_Host, pDst_Dev, total, hipMemcpyDeviceToHost), "Output Copied from Device to Host.");

	// Output the result image.
	std::cout << "Output the PGM file." << std::endl;
	WritePGM((char *)"./output/lena_after.pgm", pDst_Host, final_width, final_heigth, nMaxGray);

	getchar();
}
//  Adopted from https://github.com/komrad36/CUDALERP
__global__ void
TransformKernel(const hipTextureObject_t d_img_texA, const hipTextureObject_t d_img_texB, const hipTextureObject_t d_img_texC, 
				const float gxsA, const float gysA, 
				const float gxsB, const float gysB,
				const float gxsC, const float gysC,
				uint8_t* __restrict const d_out, const int neww) {
	uint32_t x = (blockIdx.x << 9) + (threadIdx.x << 1);
	const uint32_t y = blockIdx.y;

#pragma region Bilinear Interpolation of Image A
	const float fyA = (y + 0.5f) * gysA - 0.5f;
	const float wt_yA = fyA - floor(fyA);
	const float invwt_yA = 1.0f - wt_yA;
#pragma unroll
	for (int i = 0; i < 2; ++i, ++x) {
		const float fxA = (x + 0.5f)  * gxsA - 0.5f;
		// fA carries the d00, d01, d10, d11
		const float4 fA = tex2Dgather<float4>(d_img_texA, fxA + 0.5f, fyA + 0.5f); 
		const float wt_xA = fxA - floor(fxA); 
		const float invwt_xA = 1.0f - wt_xA;
		const float xaA = invwt_xA*fA.w + wt_xA*fA.z;
		const float xbA = invwt_xA*fA.x + wt_xA*fA.y;
		const float resA = 255.0f*(invwt_yA*xaA + wt_yA*xbA) + 0.5f;
		if (x < neww) d_out[y*neww + x] = (resA *0.34);
	}
#pragma endregion
#pragma region Bilinear Interpolation of Image B
	const float fyB = (y + 0.5f)*gysB - 0.5f;
	const float wt_yB = fyB - floor(fyB);
	const float invwt_yB = 1.0f - wt_yB;
#pragma unroll
	for (int i = 0; i < 2; ++i, ++x) {
		const float fxB = (x + 0.5f)*gxsB - 0.5f;
		const float4 fB = tex2Dgather<float4>(d_img_texB, fxB + 0.5f, fyB + 0.5f);
		const float wt_xB = fxB - floor(fxB);
		const float invwt_xB = 1.0f - wt_xB;
		const float xaB = invwt_xB*fB.w + wt_xB*fB.z;
		const float xbB = invwt_xB*fB.x + wt_xB*fB.y;
		const float resB = 255.0f*(invwt_yB*xaB + wt_yB*xbB) + 0.5f;
		if (x < neww) d_out[y*neww + x] += (resB * 0.33);
	}
#pragma endregion

#pragma region Bilinear Interpolation of Image C
	const float fyC = (y + 0.5f)*gysC - 0.5f;
	const float wt_yC = fyC - floor(fyC);
	const float invwt_yC = 1.0f - wt_yC;
#pragma unroll
	for (int i = 0; i < 2; ++i, ++x) {
		const float fxC = (x + 0.5f)*gxsC - 0.5f;
		const float4 fC = tex2Dgather<float4>(d_img_texC, fxC + 0.5f, fyC + 0.5f);
		const float wt_xC = fxC - floor(fxC);
		const float invwt_xC = 1.0f - wt_xC;
		const float xaC = invwt_xC*fC.w + wt_xC*fC.z;
		const float xbC = invwt_xC*fC.x + wt_xC*fC.y;
		const float resC = 255.0f*(invwt_yC*xaC + wt_yC*xbC) + 0.5f;
		if (x < neww) d_out[y*neww + x] += (resC * 0.33);
	}
#pragma endregion
}

void InterpolateSum(const hipTextureObject_t d_img_texA, const hipTextureObject_t d_img_texB, const hipTextureObject_t d_img_texC, const int initial_width, const int initial_height, uint8_t* __restrict const d_out, const uint32_t final_width, const uint32_t final_height) {
	const float gxsA = static_cast<float>(initial_width) / static_cast<float>(final_width);
	const float gysA = static_cast<float>(initial_height) / static_cast<float>(final_height);
	const float gxsB = static_cast<float>(initial_width / 2) / static_cast<float>(final_width);
	const float gysB = static_cast<float>(initial_height / 2) / static_cast<float>(final_height);
	const float gxsC = static_cast<float>(initial_width / 4) / static_cast<float>(final_width);
	const float gysC = static_cast<float>(initial_height / 4) / static_cast<float>(final_height);
	TransformKernel << < {((final_width - 1) >> 9) + 1, final_height}, 256 >> > (d_img_texA, d_img_texB, d_img_texC, gxsA, gysA, gxsB, gysB, gxsC, gysC, d_out, final_width);
	hipDeviceSynchronize();
}