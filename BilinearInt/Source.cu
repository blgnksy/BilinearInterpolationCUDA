#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include "npp.h"
#include <math.h>
#include <windows.h>
#include <chrono>
#include <cstdint>
#include <algorithm>
#include ""

#define VC_EXTRALEAN
#define WIN32_LEAN_AND_MEAN

// CUDA error checking Macro.
#define CUDA_CALL(x,y) {if((x) != hipSuccess){ \
  printf("CUDA error at %s:%d\n",__FILE__,__LINE__); \
  printf("  %s\n", hipGetErrorString(hipGetLastError())); \
  exit(EXIT_FAILURE);}\
  else{printf("CUDA Success at %d. (%s)\n",__LINE__,y); }}

double PCFreq = 0.0;
__int64 CounterStart = 0;

void StartCounter()
{
	LARGE_INTEGER li;
	if (!QueryPerformanceFrequency(&li))
		std::cout << "QueryPerformanceFrequency failed!\n";

	PCFreq = double(li.QuadPart) / 1000.0;

	QueryPerformanceCounter(&li);
	CounterStart = li.QuadPart;
}
double GetCounter()
{
	LARGE_INTEGER li;
	QueryPerformanceCounter(&li);
	return double(li.QuadPart - CounterStart) / PCFreq;
}

// Function Protypes.
uint8_t *
LoadPGM(char * sFileName, int & nWidth, int & nHeight, int & nMaxGray);

void
WritePGM(char * sFileName, uint8_t * pDst_Host, int nWidth, int nHeight, int nMaxGray);

__global__ void
TransformKernel(const hipTextureObject_t d_img_tex, const float gxs, const float gys, uint8_t* __restrict const d_out, const int neww);

void InterpolateSum(const hipTextureObject_t d_img_tex, const int oldw, const int oldh, uint8_t* __restrict const d_out, const uint32_t neww, const uint32_t newh);

int main()
{
	// Host parameter declarations.	
	int   nWidth, nHeight, nMaxGray, nNormalizer;

	// Load image to the host.
	std::cout << "Loading PGM file." << std::endl;
	auto pSrc_HostA = LoadPGM("lena_beforeB.pgm", nWidth, nHeight, nMaxGray);


	constexpr int oldw = 256;
	constexpr int oldh = 256;
	constexpr int neww = static_cast<int>(static_cast<double>(oldw) * 8);
	constexpr int newh = static_cast<int>(static_cast<double>(oldh) * 8);
	const size_t total = static_cast<size_t>(neww)*static_cast<size_t>(newh);


	hipDeviceSetCacheConfig(hipFuncCachePreferL1);
	hipDeviceSetSharedMemConfig(hipSharedMemBankSizeFourByte);

	hipChannelFormatDesc chandesc_img = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);

	hipArray* d_img_arr;

	CUDA_CALL(hipMallocArray(&d_img_arr, &chandesc_img, oldw, oldh, hipArrayTextureGather),"Memory Allocation.");
	CUDA_CALL(hipMemcpyToArray(d_img_arr, 0, 0, pSrc_HostA, oldh * oldw, hipMemcpyHostToDevice), "Memory Cpoied to Array.");

	struct hipResourceDesc resdesc_img;
	memset(&resdesc_img, 0, sizeof(resdesc_img));
	resdesc_img.resType = hipResourceTypeArray;
	resdesc_img.res.array.array = d_img_arr;

	struct hipTextureDesc texdesc_img;
	memset(&texdesc_img, 0, sizeof(texdesc_img));
	texdesc_img.addressMode[0] = hipAddressModeClamp;
	texdesc_img.addressMode[1] = hipAddressModeClamp;
	texdesc_img.readMode = hipReadModeNormalizedFloat;
	texdesc_img.filterMode = hipFilterModePoint;
	texdesc_img.normalizedCoords = 0;

	hipTextureObject_t d_img_tex = 0;
	CUDA_CALL(hipCreateTextureObject(&d_img_tex, &resdesc_img, &texdesc_img, nullptr),"Texture Object Created.");

	uint8_t* pDst_Dev = nullptr;
	CUDA_CALL(hipMalloc(&pDst_Dev, total),"Memory Allocated.");

	StartCounter();
	InterpolateSum(d_img_tex, oldw, oldh, pDst_Dev, neww, newh);
	std::cout << "Process finished in " << GetCounter() << std::endl;

	auto pDst_Host = new uint8_t[neww * newh];
	CUDA_CALL(hipMemcpy(pDst_Host, pDst_Dev, total, hipMemcpyDeviceToHost),"Memory Copied.");

	// Output the result image.
	std::cout << "Output the PGM file." << std::endl;
	WritePGM("lena_after.pgm", pDst_Host, neww, newh, nMaxGray);


	getchar();
}

__global__ void
TransformKernel(const hipTextureObject_t d_img_tex, const float gxs, const float gys, uint8_t* __restrict const d_out, const int neww) {
	uint32_t x = (blockIdx.x << 9) + (threadIdx.x << 1);
	const uint32_t y = blockIdx.y;
	const float fy = (y + 0.5f)*gys - 0.5f;
	const float wt_y = fy - floor(fy);
	const float invwt_y = 1.0f - wt_y;
#pragma unroll
	for (int i = 0; i < 2; ++i, ++x) {
		const float fx = (x + 0.5f)*gxs - 0.5f;
		const float4 f = tex2Dgather<float4>(d_img_tex, fx + 0.5f, fy + 0.5f);
		const float wt_x = fx - floor(fx);
		const float invwt_x = 1.0f - wt_x;
		const float xa = invwt_x*f.w + wt_x*f.z;
		const float xb = invwt_x*f.x + wt_x*f.y;
		const float res = 255.0f*(invwt_y*xa + wt_y*xb) + 0.5f;
		if (x < neww) d_out[y*neww + x] = res;
	}
}

void InterpolateSum(const hipTextureObject_t d_img_tex, const int oldw, const int oldh, uint8_t* __restrict const d_out, const uint32_t neww, const uint32_t newh) {
	const float gxs = static_cast<float>(oldw) / static_cast<float>(neww);
	const float gys = static_cast<float>(oldh) / static_cast<float>(newh);
	TransformKernel << <{((neww - 1) >> 9) + 1, newh}, 256 >> >(d_img_tex, gxs, gys, d_out, neww);
	hipDeviceSynchronize();
}

// Disable reporting warnings on functions that were marked with deprecated.
#pragma warning( disable : 4996 )

// Load PGM file.
uint8_t *
LoadPGM(char * sFileName, int & nWidth, int & nHeight, int & nMaxGray)
{
	char aLine[256];
	FILE * fInput = fopen(sFileName, "r");
	if (fInput == 0)
	{
		perror("Cannot open file to read");
		exit(EXIT_FAILURE);
	}
	// First line: version
	fgets(aLine, 256, fInput);
	std::cout << "\tVersion: " << aLine;
	// Second line: comment
	fgets(aLine, 256, fInput);
	std::cout << "\tComment: " << aLine;
	fseek(fInput, -1, SEEK_CUR);
	// Third line: size
	fscanf(fInput, "%d", &nWidth);
	std::cout << "\tWidth: " << nWidth;
	fscanf(fInput, "%d", &nHeight);
	std::cout << " Height: " << nHeight << std::endl;
	// Fourth line: max value
	fscanf(fInput, "%d", &nMaxGray);
	std::cout << "\tMax value: " << nMaxGray << std::endl;
	while (getc(fInput) != '\n');
	// Following lines: data
	uint8_t *pSrc_Host = new uint8_t[nWidth * nHeight];
	for (int i = 0; i < nHeight; ++i)
		for (int j = 0; j < nWidth; ++j)
			pSrc_Host[i*nWidth + j] = fgetc(fInput);
	fclose(fInput);

	return pSrc_Host;
}


// Write PGM image.
void
WritePGM(char *sFileName, uint8_t *pDst_Host, int nWidth, int nHeight, int nMaxGray)
{
	FILE * fOutput = fopen(sFileName, "w+");
	if (fOutput == 0)
	{
		perror("Cannot open file to read");
		exit(EXIT_FAILURE);
	}
	char * aComment = "# Created by NPP";
	fprintf(fOutput, "P5\n%s\n%d %d\n%d\n", aComment, nWidth, nHeight, nMaxGray);
	for (int i = 0; i < nHeight; ++i)
		for (int j = 0; j < nWidth; ++j)
			fputc(pDst_Host[i*nWidth + j], fOutput);
	fclose(fOutput);
}